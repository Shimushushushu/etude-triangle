
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdint.h>
#include <stdio.h>
#include <time.h>

// color_t stores r, g, and b values for output
typedef struct {
  uint8_t r, g, b;
} color_t;

// a predefined color_t which corresponds to black
__device__ const color_t kBlack = {0, 0, 0};

// vec2 stores any kind of 2-dimensional real-valued coordinates
typedef struct {
  double x, y;
} vec2;

// for clarity, uint16_t is renamed as pixel_t
typedef uint16_t pixel_t;

// the width and height of the image
const pixel_t kWidth  = 1024;
const pixel_t kHeight = 1024;

// see [^1] for details
__device__ vec2 solve(pixel_t x, pixel_t y) {
  double x_ = (double)x - 112.0;
  double y_ = (double)y - 166.0; 
  vec2 ret;
  ret.x = (691 * x_ - 399 * y_) / (800 * 691);
  ret.y = (  0 * x_ + 800 * y_) / (800 * 691);
  return ret;
}

// calculate the pixel at a pixel coordinate given by threadIdx and blockIdx
__global__ void renderer(color_t* odata) {
  uint16_t x = threadIdx.x;
  uint16_t y = blockIdx.x;

  color_t* ret = &odata[y * kWidth + x];
  vec2 tmp = solve(x, y);
  if (0 <= tmp.x && 0 <= tmp.y && tmp.x + tmp.y <= 1) {
    ret->g = tmp.x * 255;
    ret->b = tmp.y * 255;
    ret->r = (1.0 - tmp.x - tmp.y) * 255;
  } else {
    *ret = kBlack;
  }
}

int main(int argc, char** argv) {
  // allocate the canvas memory
  color_t* canvas;
  hipMalloc((void**)&canvas, kWidth * kHeight * sizeof(color_t));

  // start the timer
  struct timespec time_start = {0, 0}, time_end = {0, 0};
  clock_gettime(CLOCK_REALTIME, &time_start);

  // calculate all the pixels
  renderer<<<kWidth, kHeight>>>(canvas);

  // stop the timer, calculate the duration and print that
  clock_gettime(CLOCK_REALTIME, &time_end);
  double costTime = (time_end.tv_sec - time_start.tv_sec) * 1e9 +
      time_end.tv_nsec - time_start.tv_nsec;
  printf("duration: %.7fms\n", costTime / 1e6);

  // copy all the data back to the host
  color_t* ret = (color_t*)malloc(kWidth * kHeight * sizeof(color_t));
  hipMemcpy((void*)ret, (void*)canvas, kWidth * kHeight * sizeof(color_t),
      hipMemcpyDeviceToHost);

  // print the image in the form of ppm, coded in ASCII
  FILE* fp = fopen("out.ppm", "w");
  fprintf(fp, "P3\n%d %d\n255\n", kWidth, kHeight);
  for (size_t h = 1024; h > 0; --h) {
    for (size_t w = 0; w < 1024; ++w) {
      fprintf(fp, "%hhu %hhu %hhu\n",
          ret[(h - 1) * 1024 + w].r,
          ret[(h - 1) * 1024 + w].g,
          ret[(h - 1) * 1024 + w].b);
    }
  }
  fclose(fp);

  free(ret);
  hipFree(canvas);
  return 0;
}
